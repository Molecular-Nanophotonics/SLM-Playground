
#include "hip/hip_runtime.h"


#include <stdio.h>

__global__
void saxpy(int n, float a, float *x, float *y)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < n) y[i] = a*x[i] + y[i];
}


extern "C" __declspec(dllexport) int myFunction(int n, float a, float *x, float *y)
{
	float *d_x, *d_y;

	hipMalloc(&d_x, n*sizeof(float));
	hipMalloc(&d_y, n*sizeof(float));

	hipMemcpy(d_x, x, n*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_y, y, n*sizeof(float), hipMemcpyHostToDevice);

	// Perform SAXPY (Single-Precision A�X Plus Y) on GPU
	saxpy <<<(n + 255)/256, 256>>> (n, a, d_x, d_y);

	hipMemcpy(y, d_y, n*sizeof(float), hipMemcpyDeviceToHost);

	hipFree(d_x);
	hipFree(d_y);

	return 1;
}
